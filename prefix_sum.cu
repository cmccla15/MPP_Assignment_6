#include "hip/hip_runtime.h"
/*

*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <iostream>
#include <fstream>

using namespace std;

inline void gpu_handle_error( hipError_t err, const char* file, int line, int abort = 1 )
{
	if (err != hipSuccess)
	{
		fprintf (stderr, "gpu error %s, %s, %d\n", hipGetErrorString (err), file, line);
		if (abort)
			exit (EXIT_FAILURE);
	}
}
#define gpu_err_chk(e) {gpu_handle_error( e, __FILE__, __LINE__ );}



void scan_serial( float* scanned, float* input, int length);

//void scan_parallel( float* scanned, float* input, int length);

//__global__ void scan_parallel_kernel( ){}

int main( int argc, char** argv )
{
    ifstream input( "floats.csv" );
    string line, field;

    vector<float> floats;

    while( input.is_open() )
    {
        v.clear();
        stringstream ss( line );

        while( getline( ss, field, ',' ))
    }
}

void scan_serial( float* scanned, float* input, int length)
{
    scanned[0] = 0;
    for( int i = 1; i < length; ++i )
    {
        scanned[i] = scanned[i-1] + input[i-1];
    }
}